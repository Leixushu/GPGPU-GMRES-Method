#include "hip/hip_runtime.h"
//
//  cudaGMRES.cu
//  Cuda GMRES 
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "structs.h"
#include "parameters.h"
#include "extern.h"
#include "cuda_config.h"
#include "cuda_methods.h"
#include "cuda_dot.cu"
#include "cuda_initial2.cu"
#include "cuda_matvec.cu"
#include "cuda_matvecn.cu"
#include "cuda_vec_update.cu"
#include "cuda_reduction.cu"
#include "cuda_norm.cu"
#include "cuda_vec_replace.cu"
#include "cuda_vec_replace2.cu"
#include "cuda_matvec_up.cu"
#include "cuda_leastsq.cu"


///////////////////////////////////////////////////////
//////          SOS!!!!! O DISDIASTATOS         ///////
//////        PINAKAS u_base EINAI KATA STILI   ///////
//////          E.G. A[i][j]=A[j*ROWS+i]        ///////
///////////////////////////////////////////////////////

extern "C"
{
    //lunei to provlima A*x=r1 kai gemizei to d=x
    void cuda_GMRES( double *d, double *r1, struct common4 *sparse)
    {   
        clock_t start1, end1 ;
        float cuda_GMRES_time=0;
        int blocksPerGrid;
        blocksPerGrid=((N+threadsPerBlock-1)/threadsPerBlock);
        if (blocksPerGrid > 65530) {
            printf("WARNING,block number exceeded hardware limit");
            blocksPerGrid=65530;
        }
        //streams and devices
        hipStream_t stream0;
        hipSetDevice(0);
        hipStreamCreate( &stream0 );
        //de metraw ta apo panw sto xrono giati einai to initialization tis kartas
        printf("ThreadsPerBlock=%d\n",threadsPerBlock);
        printf("\nCuda GMRES started computation\n");
        start1 = clock(); 
        //variables declaration
        int iter=0,i=0,j=0;
        double *dev0_AA,*dev0_r1,*dev0_help;
        int *dev0_JA,*dev0_IA;
        double *dev0_x,*dev0_r0,*dev0_w,*dev0_res,*dev0_vita,*dev0_Wm;
        double *dev0_Hm,*dev0_u_base,*dev0_e,*dev0_y,*dev0_g;
    //allocation sto device arrays me dedomena
        hipMalloc((void**)&dev0_r1,(N)*sizeof(double));
        hipMalloc((void**)&dev0_AA,(Nz)*sizeof(double));
        hipMalloc((void**)&dev0_JA,(Nz)*sizeof(int));
        hipMalloc((void**)&dev0_IA,(N+1)*sizeof(int));
    //allocation sto device voithikwn arrays
        hipMalloc((void**)&dev0_x,(N)*sizeof(double));
        hipMalloc((void**)&dev0_r0,(N)*sizeof(double));
        hipMalloc((void**)&dev0_Hm,((m+1)*m)*sizeof(double));
        hipMalloc((void**)&dev0_u_base,(N*m)*sizeof(double));
        hipMalloc((void**)&dev0_e,(m+1)*sizeof(double));
        hipMalloc((void**)&dev0_y,(m)*sizeof(double));
        hipMalloc((void**)&dev0_g,(m+1)*sizeof(double));
        hipMalloc((void**)&dev0_w,(N)*sizeof(double));
        hipMalloc((void**)&dev0_res,(blocksPerGrid)*sizeof(double));
        hipMalloc((void**)&dev0_vita,sizeof(double));
        hipMalloc((void**)&dev0_Wm,((m+1)*(m+1))*sizeof(double));
        hipMalloc((void**)&dev0_help,sizeof(double));
    //perasma dedomenwn stin global device memory
        hipMemcpy(dev0_AA, sparse->AA, Nz*sizeof(double), hipMemcpyDefault );
        hipMemcpy(dev0_JA, sparse->JA, Nz*sizeof(int), hipMemcpyDefault );
        hipMemcpy(dev0_IA, sparse->IA, (N+1)*sizeof(int), hipMemcpyDefault );
        hipMemcpy(dev0_r1, r1, N*sizeof(double), hipMemcpyDefault );

/////////////////////////////////////////////////////////////////////////////////                
        //ksekinima epanaliptikis
        iter=1;
        cuda_initial2_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>((m+1),dev0_e,N,
                                                                          dev0_x);
        while (iter<=GMRES_iter) {
            //upologismos r0=b-A*x opou b=r1 me MATMUL se CSR format kai NORM r0          
            cuda_matvecn_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>(N, dev0_r0, dev0_AA, dev0_JA,
                                                                       dev0_IA, dev0_x, dev0_r1,dev0_res);
            cuda_reduction_kernel<<<1,threadsPerBlock,0,stream0>>>(blocksPerGrid, dev0_res,dev0_vita,1);
            //upologismos uj[]=r0[]/vita kai apothikeusi ston u_base KATA STILI orismenos
            //tautoxrona g=vita*e
            cuda_vec_replace_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>(  N,
                        dev0_u_base, dev0_vita, dev0_r0, m+1, dev0_g, dev0_e);    
            //KATASKEUI UPOXWROU Krylov
            for (j=0; j<m; j++) {   //j einai to count g mas
                if (j >= 1) {
                    //u_base[][j+1]=w[]/Hm[j+1][j]
                    cuda_vec_replace2_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>(N,
                                            &dev0_u_base[(j)*N], &dev0_Hm[(j)*m + j-1], dev0_w);
                }
                //uj[i]=u_base[i][j] 
                //matmul me CSR w=matvec(A,uj)
                cuda_matvec_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>( N,
                                    dev0_w, dev0_AA, dev0_JA, dev0_IA,&dev0_u_base[j*N]);  
                for (i=0; i<=j; i++) {
                    //uj[k]=u_base[k][i]
                    //DOT PRODUCT w*uj-> kai eisagwgi sto Hm[i][j]
                    cuda_dot_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>( N,dev0_w,
                                                                       &dev0_u_base[i*N],dev0_res);
                    cuda_reduction_kernel<<<1,threadsPerBlock,0,stream0>>>(blocksPerGrid, dev0_res,&dev0_Hm[i*m + j],2);   
                    //w=w-Hm(i,j)*uj
                    cuda_vec_update_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>( N,
                                                        dev0_w, &dev0_Hm[i*m + j], &dev0_u_base[i*N] );       
                }
                cuda_norm_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>(N,
                                                    dev0_w,dev0_res);
                cuda_reduction_kernel<<<1,threadsPerBlock,0,stream0>>>(blocksPerGrid, dev0_res,&dev0_Hm[(j+1)*m + j],1);
               /* if (j<(m-1)) {
                //u_base[][j+1]=w[]/Hm[j+1][j]
                    cuda_vec_replace2_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>(N,
                                        &dev0_u_base[(j+1)*N], &dev0_Hm[(j+1)*m + j], dev0_w);
                }*/
            }
            //Least Squares problem
            if (threadsPerBlock<m) {
                printf("ERROR, threadsPerBlock should be greater than m");
            }
            cuda_leastsq_kernel<<<1,threadsPerBlock,0,stream0>>>(m,dev0_Hm,dev0_g,dev0_y,dev0_Wm);
            //TELOS Least Squares problem
            //upologismos x = x0 + matvec(u_base(N,m),y(m))
            cuda_matvec_up_kernel<<<blocksPerGrid,threadsPerBlock,0,stream0>>>(N, m, dev0_x, dev0_u_base, dev0_y);
            iter++;
        }
        //Copy result back to CPU
        hipMemcpy(d, dev0_x, N*sizeof(double), hipMemcpyDefault );
        //Free memory
        printf("CUDA=%.15lf\n",d[N/2]);
        hipFree(dev0_AA);
        hipFree(dev0_JA);
        hipFree(dev0_IA);
        hipFree(dev0_r1);
        hipFree(dev0_x);
        hipFree(dev0_r0);
        hipFree(dev0_u_base);
        hipFree(dev0_Hm);
        hipFree(dev0_y);
        hipFree(dev0_g);
        hipFree(dev0_e);
        hipFree(dev0_w);
        hipFree(dev0_Wm);
        hipFree(dev0_vita);
        hipFree(dev0_res);
        end1 = clock();      
        cuda_GMRES_time = ((double) (end1 - start1)) / CLOCKS_PER_SEC;
        printf("\nXronos gia Cuda_GMRES=%.5lfs\n\n",cuda_GMRES_time);
    }
}


