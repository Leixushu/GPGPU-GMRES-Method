#include "hip/hip_runtime.h"
//
//  cuda_matvec_up.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//Matvec+vector update-> x=A*y+x opou A non-sparse matrix
__global__ void cuda_matvec_up_kernel(int Ndim,int mdim, double *x,
                                      double *A, double *y )
{
    int i=0,k=0;
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x; //global thread 
    while (thread_id < Ndim) {
        for(i = 0  ; i < Ndim*mdim ; i+=Ndim){
            x[thread_id]+= A[thread_id+i]*y[k] ; 
            k++;
        }
        thread_id += blockDim.x * gridDim.x ;
    }
} 
 
