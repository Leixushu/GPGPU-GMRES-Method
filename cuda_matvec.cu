#include "hip/hip_runtime.h"
//
//  cuda_matvec.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//
//CSR y=A*x mutliplication using CUDA
//ptr->IA       indices->JA     data->AA

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"

//PREPEI NA UPARXOUN STIN GLOBAL TIS DEVICE AA,JA,IA,x,y

//y = A*x upologizei

///////////////////////////////////////////////////////
//////       MONO GIA SPARSE MATRICES           ///////
//////       ME <32 NON-ZEROS PER ROW           ///////
///////////////////////////////////////////////////////
__global__ void cuda_matvec_kernel(int dev_dim, double *y, double *AA, int *JA, int *IA, 
                                   double *x )
{
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x; //global thread index
    int i = thread_id , jj=0;
    if (i<dev_dim) {
        while (i < dev_dim) {
            y[i] = 0;
            for( jj = IA[i] ; jj < IA[i+1]; jj ++ ){
               y[i] += AA[jj] * x[JA[jj]];
            }
            i += blockDim.x * gridDim.x;
        }
    }
}
