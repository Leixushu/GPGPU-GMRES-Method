#include "hip/hip_runtime.h"
//
//  cuda_vec_update.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//vector update y=y+ax opou y,x vectors kai a arithmos
__global__ void cuda_vec_update_kernel(int n,double *y,double *a,double *x)
{
    int global_tid=0;
    //orismos indexing
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    while (global_tid < n) {
        y[global_tid] -= ((*a)*x[global_tid]);
        global_tid += blockDim.x * gridDim.x;
    }
}
 
