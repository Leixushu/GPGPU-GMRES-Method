#include "hip/hip_runtime.h"
//
//  cuda_leastsq_kernel.cu
//  Cuda GMRES 
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"

__global__ void cuda_leastsq_kernel(int m1,double *Hm,double *g,double *y,double *W)
{
    //Serial Least Squares problem
    //Hm kai W kata grammi flattened!!!!
    //shared variables
    __shared__ double si;
    __shared__ double ci;
    double temp[threadsPerBlock],help;
    int thread_id=threadIdx.x + blockIdx.x*blockDim.x;
    int j=thread_id,p=0,q=0,i=0;
    for (i=0; i<(m1); i++) {
        j=thread_id;
        //upologismos si kai ci
        if (j==0) {
            si=Hm[(i+1)*(m1) + i]/sqrt(pow(Hm[i*(m1) + i],2) + pow(Hm[(i+1)*(m1) + i],2));
            ci=Hm[i*(m1) + i]/sqrt(pow(Hm[i*(m1) + i],2) + pow(Hm[(i+1)*(m1) + i],2));
        }
        while (j < ((m1)+1)*((m1)+1)) {
            if ((j%((m1)+2))==0) {
                W[j]=1; //stoixeia tis kurias diagwniou
            }
            else
            {
                W[j]=0;
            }
            j+=blockDim.x*gridDim.x;
        }
        __syncthreads();
        j=thread_id;
        //eisagwgi timwn pinaka stin i,i+1 grammi
        if (j==0) {
            W[i*((m1)+1)+i]=ci;
            W[i*((m1)+1)+i+1]=si;
            W[(i+1)*((m1)+1)+i]=-si;
            W[(i+1)*((m1)+1)+i+1]=ci;
        }
        __syncthreads();
        //Hm=MATMUL(W(m+1,m+1),Hm(m+1,m)
        if (j<((m1)+1)) {
            for (q=0; q < (m1); q++) {
                temp[q]=0;
                for (p=0; p<((m1)+1); p++) {
                    temp[q]+=W[j*((m1)+1)+p]*Hm[q+p*(m1)];
                }
            }
            //g=MATVEC(W,g)
            help=0;
            for(p=0;p<((m1)+1);p++){
                help+=W[j*((m1)+1)+p]*g[p];
            }
        }
        __syncthreads();
        if (j<((m1)+1)) {
            for (q=0; q < (m1); q++) {
                Hm[j*(m1)+q]=temp[q];
            }
            g[j]=help;
        }
    }
    __syncthreads();
    //Serial Back_Sub
    if (j==0) {
        for (i=(m-1);i>=0;i--){
            help=g[i];
            if (i<(m-1)){
                for (p=(i+1);p<m;p++){
                    help-=(Hm[i*m + p])*y[p];
                }
            }
            y[i]=help/(Hm[i*m + i]);
        }
        
    }
} 
    
    
    
    
    
    

