#include "hip/hip_runtime.h"
//
//  cuda_initial2.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"

//kernel initialization-> cuda_initial
//sets y=value

__global__ void cuda_initial2_kernel(int n1,double *y,int n2, double *x )
{
    int global_tid=0;
    //orismos indexing
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    while (global_tid < n1) {
        y[global_tid] = 0 ;
        if (global_tid==0) {
            y[global_tid]=1;
        }
        global_tid += blockDim.x * gridDim.x;
    }
    
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    while (global_tid < n2) {
        x[global_tid] = 0 ;
        global_tid += blockDim.x * gridDim.x;
    }
}
 



