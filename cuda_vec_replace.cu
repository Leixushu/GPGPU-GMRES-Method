#include "hip/hip_runtime.h"
//
//  cuda_vec_replace.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//vector replace y1=a*x1 opou y,x vectors kai a arithmos kai y2=x2/a
__global__ void cuda_vec_replace_kernel(int n1,double *y1 , double *a, double *x1, int n2 ,double *y2, double *x2)
{
    int global_tid=0;
    //orismos indexing
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    while (global_tid < n1) {
        y1[global_tid] = (1/(*a))*x1[global_tid];
        global_tid += blockDim.x * gridDim.x;
    }
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    while (global_tid < n2) {
        y2[global_tid] = (*a)*x2[global_tid];
        global_tid += blockDim.x * gridDim.x;
    }

}
 
 
