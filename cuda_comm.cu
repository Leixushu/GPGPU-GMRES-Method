#include "hip/hip_runtime.h"
//
//  cuda_comm.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 3/06/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//dot product dot_res=a<dot>b me diastasi dim
__global__ void cuda_comm_kernel(double *dest,double *source,int choice)
{
    if (choice==0) {
        (*dest)=sqrt((*dest)+(*source));
    }
    else
    {    
        *dest += *source;
    }
}

 
