#include "hip/hip_runtime.h"
//
//  cuda_reduction.cu
//  Cuda GMRES 
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"

__global__ void cuda_reduction_kernel(int blocks,double *dev_res, double *red_res,int choice)
{
    __shared__ double cache[threadsPerBlock]; //thread shared memory
    int j=threadIdx.x,l=0;
    int p=blocks - blockDim.x ;
    if ( p>0 ) {
        cache[j]=dev_res[j];
        l=1;
        while ((j+l*blockDim.x) < blocks) {
            cache[j]+=dev_res[j+l*blockDim.x];
            l++;
        }
    }
    else {
        if (j<blocks) {
            cache[j]=dev_res[j];
        }
        else{
            cache[j]=0;
        }
    }
    // synchronize threads in this block
    __syncthreads();
    if (blockDim.x >= 512  && threadIdx.x < 256) {
        cache[threadIdx.x] += cache[threadIdx.x + 256];
        __syncthreads();
    }
    if (blockDim.x >= 256  && threadIdx.x < 128) {
        cache[threadIdx.x] += cache[threadIdx.x + 128];
        __syncthreads();
    }
    if (blockDim.x >= 128  && threadIdx.x < 64) {
        cache[threadIdx.x] += cache[threadIdx.x + 64];
        __syncthreads();
    }
    //unroll last warp no sync needed
    if (threadIdx.x <32 ) {
        if (blockDim.x >= 64) cache[threadIdx.x] += cache[threadIdx.x +32];
        if (blockDim.x >= 32) cache[threadIdx.x] += cache[threadIdx.x +16];
        if (blockDim.x >= 16) cache[threadIdx.x] += cache[threadIdx.x +8];
        if (blockDim.x >= 8) cache[threadIdx.x] += cache[threadIdx.x +4];
        if (blockDim.x >= 4) cache[threadIdx.x] += cache[threadIdx.x +2];
        if (blockDim.x >= 2) cache[threadIdx.x] += cache[threadIdx.x +1];
    }   
    if (j==0) {
        if (choice==1) {
            *red_res=sqrt(cache[0]);
        }
        else
        {
            *red_res=cache[0];        
        }
    }
}
