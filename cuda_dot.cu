#include "hip/hip_runtime.h"
//
//  cuda_dot.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//dot product dot_res=a<dot>b me diastasi dim
__global__ void cuda_dot_kernel(int n,double *a, double *b, double *dot_res)
{

    __shared__ double cache[threadsPerBlock]; //thread shared memory
    int global_tid=0,cacheIndex=0;
    double temp = 0; 
    //orismos indexing
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    cacheIndex = threadIdx.x;
    while (global_tid < n) {
        temp += a[global_tid] * b[global_tid];
        global_tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    // synchronize threads in this block
    __syncthreads();
    if (blockDim.x >= 1024  && threadIdx.x < 512) {
        cache[threadIdx.x] += cache[threadIdx.x + 512];
        __syncthreads();
    }
    if (blockDim.x >= 512  && threadIdx.x < 256) {
        cache[threadIdx.x] += cache[threadIdx.x + 256];
        __syncthreads();
    }
    if (blockDim.x >= 256  && threadIdx.x < 128) {
        cache[threadIdx.x] += cache[threadIdx.x + 128];
        __syncthreads();
    }
    if (blockDim.x >= 128  && threadIdx.x < 64) {
        cache[threadIdx.x] += cache[threadIdx.x + 64];
        __syncthreads();
    }
    //unroll last warp no sync needed
    if (threadIdx.x <32 ) {
        if (blockDim.x >= 64) cache[threadIdx.x] += cache[threadIdx.x +32];
        if (blockDim.x >= 32) cache[threadIdx.x] += cache[threadIdx.x +16];
        if (blockDim.x >= 16) cache[threadIdx.x] += cache[threadIdx.x +8];
        if (blockDim.x >= 8) cache[threadIdx.x] += cache[threadIdx.x +4];
        if (blockDim.x >= 4) cache[threadIdx.x] += cache[threadIdx.x +2];
        if (blockDim.x >= 2) cache[threadIdx.x] += cache[threadIdx.x +1];
    }   

    if (cacheIndex==0) {
        dot_res[blockIdx.x]=cache[0];
    }
   
}

 
