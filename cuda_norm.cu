#include "hip/hip_runtime.h"
//
//  cuda_norm.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//euclidian norm of vector vec
__global__ void cuda_norm_kernel(int n,double *a, double *dot_res)
{
    
    __shared__ double cache[threadsPerBlock]; //thread shared memory
    int thread_id=0,cacheIndex=0;
    double temp1 = 0; 
    //orismos indexing
    thread_id = threadIdx.x + blockIdx.x * blockDim.x; 
    cacheIndex = threadIdx.x;
    while (thread_id < n) {
        temp1 += a[thread_id] * a[thread_id];
        thread_id += blockDim.x * gridDim.x;
    }
    // set the cache values
    cache[cacheIndex] = temp1;
    // synchronize threads in this block
    __syncthreads();
    if (blockDim.x >= 512  && threadIdx.x < 256) {
        cache[threadIdx.x] += cache[threadIdx.x + 256];
        __syncthreads();
    }
    if (blockDim.x >= 256  && threadIdx.x < 128) {
        cache[threadIdx.x] += cache[threadIdx.x + 128];
        __syncthreads();
    }
    if (blockDim.x >= 128  && threadIdx.x < 64) {
        cache[threadIdx.x] += cache[threadIdx.x + 64];
        __syncthreads();
    }
    //unroll last warp no sync needed
    if (threadIdx.x <32 ) {
        if (blockDim.x >= 64) cache[threadIdx.x] += cache[threadIdx.x +32];
        if (blockDim.x >= 32) cache[threadIdx.x] += cache[threadIdx.x +16];
        if (blockDim.x >= 16) cache[threadIdx.x] += cache[threadIdx.x +8];
        if (blockDim.x >= 8) cache[threadIdx.x] += cache[threadIdx.x +4];
        if (blockDim.x >= 4) cache[threadIdx.x] += cache[threadIdx.x +2];
        if (blockDim.x >= 2) cache[threadIdx.x] += cache[threadIdx.x +1];
    }   
    if (cacheIndex==0) {
        dot_res[blockIdx.x]=cache[0];
    }
}
