#include "hip/hip_runtime.h"
//
//  cuda_matvecn.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//
//CSR y=A*x mutliplication using CUDA
//ptr->IA       indices->JA     data->AA

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"

//PREPEI NA UPARXOUN STIN GLOBAL TIS DEVICE AA,JA,IA,x,y

//y = b - A*x upologizei

///////////////////////////////////////////////////////
//////       MONO GIA SPARSE MATRICES           ///////
//////       ME <32 NON-ZEROS PER ROW           ///////
///////////////////////////////////////////////////////
__global__ void cuda_matvecn_kernel(int dev_dim, double *y,double *AA, int *JA,
                                    int *IA, double *x, double *b,double *dot_res)

{
    __shared__ double cache[threadsPerBlock]; //thread shared memory
    int thread_id=threadIdx.x+blockIdx.x*blockDim.x; //global thread index
    int cacheIndex=threadIdx.x;
    double temp1 = 0;
    double temp2 = 0;
    int i = thread_id , jj=0;
    if (i<(dev_dim)) {
        while (i < (dev_dim)) {
            temp1 = 0;
            temp2 = 0;
            for( jj = IA[i] ; jj < IA[i+1]; jj ++ ){
                temp1 += AA[jj] * x[JA[jj]];
            }
            y[i] = b[i]-temp1;
            temp2 += y[i] * y[i];
            i += blockDim.x * gridDim.x;
        }
    }
    // set the cache values
    cache[cacheIndex] = temp2;
    // synchronize threads in this block
    __syncthreads();
    if (blockDim.x >= 512  && threadIdx.x < 256) {
        cache[threadIdx.x] += cache[threadIdx.x + 256];
        __syncthreads();
    }
    if (blockDim.x >= 256  && threadIdx.x < 128) {
        cache[threadIdx.x] += cache[threadIdx.x + 128];
        __syncthreads();
    }
    if (blockDim.x >= 128  && threadIdx.x < 64) {
        cache[threadIdx.x] += cache[threadIdx.x + 64];
        __syncthreads();
    }
    //unroll last warp no sync needed
    if (threadIdx.x <32 ) {
        if (blockDim.x >= 64) cache[threadIdx.x] += cache[threadIdx.x +32];
        if (blockDim.x >= 32) cache[threadIdx.x] += cache[threadIdx.x +16];
        if (blockDim.x >= 16) cache[threadIdx.x] += cache[threadIdx.x +8];
        if (blockDim.x >= 8) cache[threadIdx.x] += cache[threadIdx.x +4];
        if (blockDim.x >= 4) cache[threadIdx.x] += cache[threadIdx.x +2];
        if (blockDim.x >= 2) cache[threadIdx.x] += cache[threadIdx.x +1];
    }   
    if (cacheIndex==0) {
        dot_res[blockIdx.x]=cache[0];
    }
}

