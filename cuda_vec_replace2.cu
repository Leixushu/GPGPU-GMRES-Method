#include "hip/hip_runtime.h"
//
//  cuda_vec_replace2.cu
//  Cuda GMRES
//
//  Created by Tim Ioannidis on 2/18/12.
//  Copyright 2012 Chemeng NTUA. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cuda_config.h"
#include "cuda_methods.h"


//vector replace y1=a*x1 opou y,x vectors kai a arithmos kai y2=x2/a
__global__ void cuda_vec_replace2_kernel(int n1,double *y1 , double *a,
                      double *x1 )
{

    int global_tid=0;
    //orismos indexing
    global_tid = threadIdx.x + blockIdx.x * blockDim.x; 
    while (global_tid < n1) {
        y1[global_tid] = (1/(*a))*x1[global_tid];
        global_tid += blockDim.x * gridDim.x;
    }
}
 
 
